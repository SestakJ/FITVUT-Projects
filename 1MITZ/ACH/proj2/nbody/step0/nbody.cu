#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xbolva00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) {
		float tx, ty, tz;
		tx = ty = tz = 0.0f;
		float g_dt_inv = G * dt;
	        float p_pos_x_i = p.pos[i].x;
	        float p_pos_y_i = p.pos[i].y;
	        float p_pos_z_i = p.pos[i].z;
		for (int j = 0; j < N; ++j) {
			float dx = p.pos[j].x - p_pos_x_i;
			float dy = p.pos[j].y - p_pos_y_i;
			float dz = p.pos[j].z - p_pos_z_i;
			float s = dx*dx + dy*dy + dz*dz;
			float r = sqrtf(s);
			bool ok = r > COLLISION_DISTANCE;
			float vel = (ok) ? (g_dt_inv * p.pos[j].w) / (s * r)  : 0.0f;
			
			tx += (ok) ? dx * vel : 0.0f;
			ty += (ok) ? dy * vel : 0.0f;
			tz += (ok) ? dz * vel : 0.0f;
		}

		tmp_vel.coord[i].x += tx;
		tmp_vel.coord[i].y += ty;
		tmp_vel.coord[i].z += tz;

   }
}

__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) {
		float tx, ty, tz;
		tx = ty = tz = 0.0f;
	        float p_w_i = p.pos[i].w;
	        float p_pos_x_i = p.pos[i].x;
	        float p_pos_y_i = p.pos[i].y;
	        float p_pos_z_i = p.pos[i].z;
	        float p_vel_x_i = p.vel[i].x;
	        float p_vel_y_i = p.vel[i].y;
	        float p_vel_z_i = p.vel[i].z;
		for (int j = 0; j < N; ++j) {
			float dx = p_pos_x_i - p.pos[j].x;
			float dy = p_pos_y_i - p.pos[j].y;
			float dz = p_pos_z_i - p.pos[j].z;
			float s = dx*dx + dy*dy + dz*dz;
			float r = sqrtf(s);
			
			float p_w_j = p.pos[j].w;
			float subw = p_w_i - p_w_j;
			float addw = p_w_i + p_w_j;
			float p_2w_j = 2 * p_w_j;
			bool ok = r > 0.0f && r < COLLISION_DISTANCE;

			tx += (ok) ? ((p_2w_j * p.vel[j].x + subw * p_vel_x_i) / addw) - p_vel_x_i : 0.0f;
			ty += (ok) ? ((p_2w_j * p.vel[j].y + subw * p_vel_y_i) / addw) - p_vel_y_i : 0.0f;
			tz += (ok) ? ((p_2w_j * p.vel[j].z + subw * p_vel_z_i) / addw) - p_vel_z_i : 0.0f;
		}

		tmp_vel.coord[i].x += tx;
		tmp_vel.coord[i].y += ty;
		tmp_vel.coord[i].z += tz;

	}
}

__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) {
		p.vel[i].x += tmp_vel.coord[i].x;
		p.vel[i].y += tmp_vel.coord[i].y;
		p.vel[i].z += tmp_vel.coord[i].z;

		p.pos[i].x += p.vel[i].x * dt;
		p.pos[i].y += p.vel[i].y * dt;
		p.pos[i].z += p.vel[i].z * dt;

		tmp_vel.coord[i].x = tmp_vel.coord[i].y = tmp_vel.coord[i].z = 0.0f;
   }
}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++)
	{
		fscanf(fp, "%f %f %f %f %f %f %f \n",
				&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
				&p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
				&p.pos[i].w);
	}
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++)
	{
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
				p.pos[i].x, p.pos[i].y, p.pos[i].z,
				p.vel[i].x, p.vel[i].y, p.vel[i].z,
				p.pos[i].w);
	}
}
