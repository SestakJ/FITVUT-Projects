#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xbolva00
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <algorithm>

#include "nbody.h"

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

    // alokace pameti na CPU
    t_particles particles_cpu;
    const size_t N_float4 = N * sizeof(float4);
    hipHostAlloc(&particles_cpu.pos, N_float4, hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel, N_float4, hipHostMallocDefault);

    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[5]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);

    t_particles particles_gpu;
    t_particles particles2_gpu;

    hipMalloc(&particles_gpu.pos, N_float4);
    hipMalloc(&particles_gpu.vel, N_float4);
    hipMalloc(&particles2_gpu.pos, N_float4);
    hipMalloc(&particles2_gpu.vel, N_float4);

    hipMemcpy(particles_gpu.pos, particles_cpu.pos, N_float4, hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.vel, particles_cpu.vel, N_float4, hipMemcpyHostToDevice);
    hipMemcpy(particles2_gpu.pos, particles_cpu.pos, N_float4, hipMemcpyHostToDevice);
    hipMemcpy(particles2_gpu.vel, particles_cpu.vel, N_float4, hipMemcpyHostToDevice);

    size_t grid = (N + thr_blc - 1) / thr_blc;
    
    // vypocet
    gettimeofday(&t1, 0);

    for (int s = 0; s < steps; ++s)
    {
        calculate_velocity<<<grid, thr_blc>>>(particles_gpu, particles2_gpu, N, dt);
	std::swap(particles_gpu, particles2_gpu);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    hipMemcpy(particles_cpu.pos, particles_gpu.pos, N_float4, hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel, particles_gpu.vel, N_float4, hipMemcpyDeviceToHost);

    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);

    return 0;
}
