#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xbolva00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_velocity(t_particles p_in, t_particles p_out, const int N, const float dt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
	        float tx = 0.0f, ty = 0.0f, tz = 0.0f;
		float g_dt_inv = G * dt;
	        float p_w_i = p_in.pos[i].w;
	        float p_pos_x_i = p_in.pos[i].x;
	        float p_pos_y_i = p_in.pos[i].y;
	        float p_pos_z_i = p_in.pos[i].z;
	        float p_vel_x_i = p_in.vel[i].x;
	        float p_vel_y_i = p_in.vel[i].y;
	        float p_vel_z_i = p_in.vel[i].z;

		for (int j = 0; j < N; ++j) {
			float dx = p_in.pos[j].x - p_pos_x_i;
			float dy = p_in.pos[j].y - p_pos_y_i;
			float dz = p_in.pos[j].z - p_pos_z_i;

			float s = dx * dx + dy * dy + dz * dz;
			float r = sqrtf(s);

		 	float p_w_j = p_in.pos[j].w;
			float sub_w = p_w_i - p_w_j;
			float add_w = p_w_i + p_w_j;
			float v_j = (r > COLLISION_DISTANCE) ? (g_dt_inv * p_w_j) / (s * r) : 0.0f;

			tx += (r > COLLISION_DISTANCE) ? (dx * v_j) : ((2 * p_w_j * p_in.vel[j].x + sub_w * p_vel_x_i) / add_w) - p_vel_x_i;
			ty += (r > COLLISION_DISTANCE) ? (dy * v_j) : ((2 * p_w_j * p_in.vel[j].y + sub_w * p_vel_y_i) / add_w) - p_vel_y_i;
			tz += (r > COLLISION_DISTANCE) ? (dz * v_j) : ((2 * p_w_j * p_in.vel[j].z + sub_w * p_vel_z_i) / add_w) - p_vel_z_i;

		}

		p_out.vel[i].x = p_vel_x_i + tx;
		p_out.vel[i].y = p_vel_y_i + ty;
		p_out.vel[i].z = p_vel_z_i + tz;

		p_out.pos[i].x = p_out.vel[i].x * dt + p_pos_x_i;
		p_out.pos[i].y = p_out.vel[i].y * dt + p_pos_y_i;
		p_out.pos[i].z = p_out.vel[i].z * dt + p_pos_z_i;

	}
}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++)
	{
		fscanf(fp, "%f %f %f %f %f %f %f \n",
				&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
				&p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
				&p.pos[i].w);
	}
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++)
	{
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
				p.pos[i].x, p.pos[i].y, p.pos[i].z,
				p.vel[i].x, p.vel[i].y, p.vel[i].z,
				p.pos[i].w);
	}
}
