#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2018)
 * Projekt c.2 (CUDA)
 * Login: xbolva00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_velocity(t_particles p_in, t_particles p_out, const int N, const float dt)
{
	extern __shared__ float4 sh_mem_pos[];

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	float tx = 0.0f, ty = 0.0f, tz = 0.0f;
	float g_dt_inv = G * dt;
	float p_w_i = p_in.pos[i].w;
	float p_pos_x_i = p_in.pos[i].x;
	float p_pos_y_i = p_in.pos[i].y;
	float p_pos_z_i = p_in.pos[i].z;
	float p_vel_x_i = p_in.vel[i].x;
	float p_vel_y_i = p_in.vel[i].y;
	float p_vel_z_i = p_in.vel[i].z;
		
	for (int tile = 0; tile < gridDim.x; ++tile) {
		int base = tile * blockDim.x;
		int idx = base + threadIdx.x; 
		if (idx < N) 			
			sh_mem_pos[threadIdx.x] = p_in.pos[idx];
		__syncthreads();
	
		for (int j = 0; j < blockDim.x; ++j) {
			if (base + j >= N) break; 
			float dx = sh_mem_pos[j].x - p_pos_x_i;
			float dy = sh_mem_pos[j].y - p_pos_y_i;
			float dz = sh_mem_pos[j].z - p_pos_z_i;

			float s = dx * dx + dy * dy + dz * dz;
			float r = sqrtf(s);

			float p_w_j = sh_mem_pos[j].w;
			float sub_w = p_w_i - p_w_j;
			float add_w = p_w_i + p_w_j;
			float v_j = (r > COLLISION_DISTANCE) ? (g_dt_inv * p_w_j) / (s * r) : 0.0f;

			tx += (r > COLLISION_DISTANCE) ? (dx * v_j) : ((2 * p_w_j * p_in.vel[tile * blockDim.x + j].x + sub_w * p_vel_x_i) / add_w) - p_vel_x_i;
			ty += (r > COLLISION_DISTANCE) ? (dy * v_j) : ((2 * p_w_j * p_in.vel[tile * blockDim.x + j].y + sub_w * p_vel_y_i) / add_w) - p_vel_y_i;
			tz += (r > COLLISION_DISTANCE) ? (dz * v_j) : ((2 * p_w_j * p_in.vel[tile * blockDim.x + j].z + sub_w * p_vel_z_i) / add_w) - p_vel_z_i;
	
		}
		
		__syncthreads();
	}
		
	if (i < N) {
		p_out.vel[i].x = p_vel_x_i + tx;
		p_out.vel[i].y = p_vel_y_i + ty;
		p_out.vel[i].z = p_vel_z_i + tz;

		p_out.pos[i].x = p_out.vel[i].x * dt + p_pos_x_i;
		p_out.pos[i].y = p_out.vel[i].y * dt + p_pos_y_i;
		p_out.pos[i].z = p_out.vel[i].z * dt + p_pos_z_i;
	}
}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++)
	{
		fscanf(fp, "%f %f %f %f %f %f %f \n",
				&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
				&p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
				&p.pos[i].w);
	}
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
  for (int i = 0; i < N; i++)
	{
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
				p.pos[i].x, p.pos[i].y, p.pos[i].z,
				p.vel[i].x, p.vel[i].y, p.vel[i].z,
				p.pos[i].w);
	}
}
